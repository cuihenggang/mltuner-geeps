#include "hip/hip_runtime.h"
#ifndef CPU_ONLY

#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>
#include <cstdlib>
#include <cstring>

#include "math_functions.hpp"

template <>
void gpu_gemm<float>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(cublas_handle, cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gpu_gemm<double>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(cublas_handle, cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void gpu_gemm<float>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const int lda,
    const float* B, const int ldb,
    const float beta, float* C, const int ldc) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(cublas_handle, cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void gpu_gemm<double>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const int lda,
    const double* B, const int ldb,
    const double beta, double* C, const int ldc) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(cublas_handle, cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, ldc));
}

template <>
void gpu_gemv<float>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(cublas_handle, cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void gpu_gemv<double>(
    hipblasHandle_t cublas_handle, const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(cublas_handle, cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void gpu_axpy<float>(
    hipblasHandle_t cublas_handle, const int N,
    const float alpha, const float* X, float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}

template <>
void gpu_axpy<double>(
    hipblasHandle_t cublas_handle, const int N,
    const double alpha, const double* X, double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}

template <>
float gpu_dot<float>(hipblasHandle_t cublas_handle,
    const int n, const float* x, const float* y) {
  float out;
  CUBLAS_CHECK(hipblasSdot(cublas_handle, n, x, 1, y, 1, &out));
  return out;
}

template <>
double gpu_dot<double>(hipblasHandle_t cublas_handle,
    const int n, const double* x, const double* y) {
  double out;
  CUBLAS_CHECK(hipblasDdot(cublas_handle, n, x, 1, y, 1, &out));
  return out;
}

template <>
float gpu_asum<float>(
    hipblasHandle_t cublas_handle, const int n, const float* x) {
  float y;
  CUBLAS_CHECK(hipblasSasum(cublas_handle, n, x, 1, &y));
  return y;
}

template <>
double gpu_asum<double>(
    hipblasHandle_t cublas_handle, const int n, const double* x) {
  double y;
  CUBLAS_CHECK(hipblasDasum(cublas_handle, n, x, 1, &y));
  return y;
}

#endif