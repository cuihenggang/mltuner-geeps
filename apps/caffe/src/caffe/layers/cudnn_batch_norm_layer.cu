
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layers/cudnn_batch_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  CHECK_EQ(this->blobs_.size(), 2);
  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  const Dtype* bias_data = this->blobs_[1]->gpu_data();

  CHECK_EQ(top.size(), 5);
  Dtype* top_data = top[0]->mutable_gpu_data();
  // Dtype* save_mean = save_mean_.mutable_gpu_data();
  // Dtype* save_inv_var = save_inv_var_.mutable_gpu_data();
  Dtype* save_mean = top[1]->mutable_gpu_data();
  Dtype* save_inv_var = top[2]->mutable_gpu_data();
  double epsilon = max(this->eps_, CUDNN_BN_MIN_EPSILON);

  /* Cui: We want to use the cudnnBatchNormalizationForwardTraining() method
   * for both training and testing, because I don't want to use
   * the running_mean and running_var. */
  /* Cui TODO: since we are not using running_mean and running_var,
   * top[3] and top[4] can be removed */
  // if (this->phase_ == TRAIN) {
  if (true) {
    Dtype* running_mean = top[3]->mutable_gpu_data();
    Dtype* running_var = top[4]->mutable_gpu_data();
    // Call Batch normalization forward
    CUDNN_CHECK(cudnnBatchNormalizationForwardTraining(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      1-this->moving_average_fraction_,
      running_mean,  // mean
      running_var,  // variance
      epsilon,
      save_mean,
      save_inv_var));
  } else if (this->phase_ == TEST) {
    const Dtype* running_mean = top[3]->gpu_data();
    const Dtype* running_var = top[4]->gpu_data();
    CUDNN_CHECK(cudnnBatchNormalizationForwardInference(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_data,
      scale_bias_mean_var_desc_,
      scale_data,
      bias_data,
      running_mean,  // mean
      running_var,  // variance
      epsilon));
  } else {
    LOG(FATAL) << "Unknown phase";
  }
}

template <typename Dtype>
void CuDNNBatchNormLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  // const Dtype* save_mean = save_mean_.gpu_data();
  // const Dtype* save_inv_var = save_inv_var_.gpu_data();
  const Dtype* save_mean = top[1]->gpu_data();
  const Dtype* save_inv_var = top[2]->gpu_data();

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* scale_data = this->blobs_[0]->gpu_data();
  Dtype* scale_diff = this->blobs_[0]->mutable_gpu_diff();
  Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();

  double epsilon = max(this->eps_, CUDNN_BN_MIN_EPSILON);

  // call Batch Normalization Backward
  CUDNN_CHECK(cudnnBatchNormalizationBackward(
      Caffe::cudnn_handle(),
      mode_,
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::zero,
#if CUDNN_VERSION >= 4005
      cudnn::dataType<Dtype>::one,
      cudnn::dataType<Dtype>::one,
#endif
      bottom_desc_,
      bottom_data,
      bottom_desc_,
      top_diff,
      bottom_desc_,
      bottom_diff,
      scale_bias_mean_var_desc_,
      scale_data,
      scale_diff,
      bias_diff,
      epsilon,
      save_mean,
      save_inv_var));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNBatchNormLayer);

}  // namespace caffe
#endif
